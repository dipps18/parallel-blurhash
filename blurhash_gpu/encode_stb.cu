#include "encode.cuh"

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#include <stdio.h>

const char *blurHashForFile(int xComponents, int yComponents,const char *filename);

int main(int argc, const char **argv) {
	if(argc != 4) {
		fprintf(stderr, "Usage: %s x_components y_components imagefile\n", argv[0]);
		return 1;
	}

	int xComponents = atoi(argv[1]);
	int yComponents = atoi(argv[2]);
	if(xComponents < 1 || xComponents > 8 || yComponents < 1 || yComponents > 8) {
		fprintf(stderr, "Component counts must be between 1 and 8.\n");
		return 1;
	}

	// int xComponents = 3;
	// int yComponents = 3;
	// char file[26] = {0};
	// strcpy(file, "../images/man_sitting.jpg");
	const char *hash = blurHashForFile(xComponents, yComponents, argv[3]);
	//const char *hash = blurHashForFile(xComponents, yComponents, file);
	if(!hash) {
		fprintf(stderr, "Failed to load image file \"%s\".\n", argv[3]);
		return 1;
	}

	printf("%s\n", hash);

	return 0;
}

const char *blurHashForFile(int xComponents, int yComponents,const char *filename) {
	int width, height, channels;

	unsigned char *data = stbi_load(filename, &width, &height, &channels, 3);
	if(!data) return NULL;
	hipEvent_t start, stop;
	float time = 0;
	hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
	const char *hash = blurHashForPixels(xComponents, yComponents, width, height, data, width * 3);
	hipEventRecord(stop);
    hipEventElapsedTime(&time, start, stop);
	printf("Time taken for algorithm in GPU: %fms\n", time);
	stbi_image_free(data);
	return hash;
}